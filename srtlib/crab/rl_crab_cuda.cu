#include "hip/hip_runtime.h"
#include<unistd.h>
#include "mir_math.h"
#include "rl_crab_cuda.h"
#include <hip/hip_runtime.h>
#include "hipblas.h"

__global__
void SrtlibRlCrabCuda::VecDiv(
    const double* const vec1_arr,
    const double* const vec2_arr,
    int nsize,
    double* const vec3_arr)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < nsize){
        vec3_arr[index] = vec1_arr[index] / vec2_arr[index];
    }
    __syncthreads();
}

__global__
void SrtlibRlCrabCuda::VecMul(
    const double* const vec1_arr,
    const double* const vec2_arr,
    int nsize,
    double* const vec3_arr)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if(index < nsize){    
        vec3_arr[index] = vec1_arr[index] * vec2_arr[index];
    }
    __syncthreads();
}

void SrtlibRlCrabCuda::GetDetArr(
    hipblasHandle_t handle,
    const double* const sky_dev_arr,
    const double* const resp_norm_mat_dev_arr,
    int ndet, int nsky,
    double* const det_dev_arr) // ndet
{
    double alpha = 1.0;
    double beta = 0.0;
    hipblasDgemv(handle, HIPBLAS_OP_N,
		ndet, nsky, &alpha,
		resp_norm_mat_dev_arr, ndet,
		sky_dev_arr, 1,
		&beta, det_dev_arr, 1);
}

void SrtlibRlCrabCuda::GetDenArr(
    hipblasHandle_t handle,
    const double* const sky_dev_arr,
    const double* const flux_dev_arr,
    const double* const det_0_dev_arr,
    const double* const bg_dev_arr,
    const double* const resp_norm_mat_dev_arr,
    int ndet, int nsky, int nphase,
    double* const* const den_dev_arr)
{
    double* det_dev_arr = NULL;
    size_t mem_size_ndet = ndet * sizeof(double);
    hipMalloc((void **)&det_dev_arr, mem_size_ndet);
    SrtlibRlCrabCuda::GetDetArr(handle, sky_dev_arr,
                                resp_norm_mat_dev_arr,
                                ndet, nsky, det_dev_arr);
    double* flux_arr = new double[nphase];
    hipblasGetVector(nphase, sizeof(double), flux_dev_arr, 1,
                    flux_arr, 1);
    for(int iphase = 0; iphase < nphase; iphase++){
        double flux_tmp = flux_arr[iphase];
        hipblasDcopy(handle, ndet,
                    bg_dev_arr, 1,
                    den_dev_arr[iphase], 1);
        hipblasDaxpy(handle, ndet,
                    &flux_tmp,
                    det_0_dev_arr, 1,
                    den_dev_arr[iphase], 1);
        double alpha = 1.0;
        hipblasDaxpy(handle, ndet,
                    &alpha, det_dev_arr, 1,
                    den_dev_arr[iphase], 1);
    }
    delete [] flux_arr;
    hipFree(det_dev_arr);
}


void SrtlibRlCrabCuda::GetYDashArr(
    const double* const* const data_dev_arr,
    const double* const* const den_dev_arr,
    int ndet, int nphase,
    double* const* const y_dash_dev_arr)
{
    int blocksize = 512;
    dim3 block (blocksize, 1, 1);
    dim3 grid (ndet / block.x + 1, 1, 1);
    for(int iphase = 0; iphase < nphase; iphase++){
        SrtlibRlCrabCuda::VecDiv<<<grid,block>>>(
            data_dev_arr[iphase], den_dev_arr[iphase],
            ndet, y_dash_dev_arr[iphase]);
    }
}

void SrtlibRlCrabCuda::GetMvalArr(
    hipblasHandle_t handle,
    const double* const* const y_dash_dev_arr,
    const double* const resp_norm_mat_dev_arr,
    const double* const sky_dev_arr,
    int ndet, int nsky, int nphase,
    double* const mval_dev_arr)
{
    double* y_dash_sum_dev_arr = NULL;
    size_t mem_size_ndet = ndet * sizeof(double);
    hipMalloc((void **)&y_dash_sum_dev_arr, mem_size_ndet);
    hipblasDcopy(handle, ndet,
                y_dash_dev_arr[0], 1,
                y_dash_sum_dev_arr, 1);
    for(int iphase = 1; iphase < nphase; iphase++){
        double alpha = 1.0;
        hipblasDaxpy(handle, ndet,
                    &alpha, y_dash_dev_arr[iphase], 1,
                    y_dash_sum_dev_arr, 1);
    }
    double* coeff_dev_arr = NULL;
    size_t mem_size_nsky = nsky * sizeof(double);
    hipMalloc((void **)&coeff_dev_arr, mem_size_nsky);
    double alpha = 1.0;
    double beta = 0.0;
    hipblasDgemv(handle, HIPBLAS_OP_T,
                ndet, nsky, &alpha,
                resp_norm_mat_dev_arr, ndet,
                y_dash_sum_dev_arr, 1,
                &beta, coeff_dev_arr, 1);
    int blocksize = 512;
    dim3 block (blocksize, 1, 1);
    dim3 grid (nsky / block.x + 1, 1, 1);
    SrtlibRlCrabCuda::VecMul<<<grid,block>>>(
        coeff_dev_arr, sky_dev_arr, nsky, mval_dev_arr);
    hipFree(y_dash_sum_dev_arr);
    hipFree(coeff_dev_arr);
}

void SrtlibRlCrabCuda::GetNvalArr(
    hipblasHandle_t handle,
    const double* const* const y_dash_dev_arr,
    const double* const flux_dev_arr,
    const double* const det_0_dev_arr,
    int ndet, int nphase,
    double* const nval_dev_arr)
{
    double* dot_arr = new double[nphase];
    for(int iphase = 0; iphase < nphase; iphase++){
        double dot = 0.0;
        hipblasDdot(handle, ndet,
                   y_dash_dev_arr[iphase], 1,
                   det_0_dev_arr, 1, &dot);
        dot_arr[iphase] = dot;
    }
    double* dot_dev_arr = NULL;
    size_t mem_size_nphase = nphase * sizeof(double);
    hipMalloc((void **)&dot_dev_arr, mem_size_nphase);
    hipblasSetVector(nphase, sizeof(double), dot_arr, 1,
		    dot_dev_arr, 1);
    int blocksize = 512;
    dim3 block (blocksize, 1, 1);
    dim3 grid (nphase / block.x + 1, 1, 1);
    SrtlibRlCrabCuda::VecMul<<<grid,block>>>(flux_dev_arr,
                                             dot_dev_arr,
                                             nphase,
                                             nval_dev_arr);
    delete [] dot_arr;
    hipFree(dot_dev_arr);
}
